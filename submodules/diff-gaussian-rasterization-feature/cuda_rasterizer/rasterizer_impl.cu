#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"




// Helper function to find the next-highest bit of the MSB
// on the CPU.
// 查找most significant bit, 輸入變量n表示tile編號最大值x,y的乘積
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}





// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}



// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets, // inclusive sum of numbr of rendered gaussians
	uint64_t* gaussian_keys_unsorted, // outpuit
	uint32_t* gaussian_values_unsorted, //output
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
// 針對每一個tile決定其 key list 的開始與結束位置。
// L num rendered
// 決定point_list_keys 在哪一個tile開始與結束
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}






// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum <<<(P + 255) / 256, 256 >>> (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}







CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P) // P gaussians number
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128); // radius pointer
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.semantic_feature, P * NUM_SEMANTIC_CHANNELS, 128);
	obtain(chunk, geom.score_feature, P , 128);
	obtain(chunk, geom.tiles_touched, P, 128); // record the touched tiles for each gaussian
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128); // tmp for Inclusivesum
	obtain(chunk, geom.point_offsets, P, 128); // inclusive sum offset for tiles touched
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N) // N: image pixels width * height
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128); // tile_grid.x * tile_grid.y * sizeof(uint2))
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P) // P num_rendered tiles
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128); // value: gaussian id
	obtain(chunk, binning.point_list_unsorted, P, 128); // vlaue: gaussian id
	obtain(chunk, binning.point_list_keys, P, 128); // key: tile|depth
	obtain(chunk, binning.point_list_keys_unsorted, P, 128); //key: tile:depth
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128); // tmp sorting space
	return binning;
}







// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* semantic_feature, 
	const float* score_feature, ////
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color, // output 
	float* out_feature_map, // output 
	float* out_score_map, // output 
	float* out_depth, // output 
	int* radii, // output 
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);


	// create geonetry state
	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P); 


	// make radius
	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}


	// create block, tile_grid
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);
	
	
	// Dynamically resize image-based auxiliary buffers during training
	// create image state
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	
	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}


	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii, // o radius
		geomState.means2D, //o xy coords
		geomState.depths, //o 
		geomState.cov3D, //o 
		geomState.rgb, //o  2d means color
		geomState.conic_opacity,// output conic->used for compute prob
		tile_grid, // tile total number (x,y)
		geomState.tiles_touched, // o tiles touched sum array
		prefiltered
	), debug)


	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)


	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered; // total rendered number in pixels
	// geomState.point_offsets 存每個gaussians碰到的tile總數
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);


	// create bining state
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);


	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	// P: number of gaussians
	duplicateWithKeys <<<(P + 255) / 256, 256 >>> (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets, // inclusive sum for rendered gaussians
		binningState.point_list_keys_unsorted, // o
		binningState.point_list_unsorted, // o
		radii,
		tile_grid
	)
	CHECK_CUDA(, debug)

	// point_list_keys_unsorted被tile總數所限制
	int bit = getHigherMsb(tile_grid.x * tile_grid.y); // get the fighest bit position


	// Sort complete list of (duplicated) Gaussian indices by keys
	// sort gaussian total list
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)


	// 製作一個array, tile_x* tile_y* 2, 紀錄每個tile在point lsit中的開始與結束位置
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);



	// Identify start and end of per-tile workloads in sorted list
	// 紀錄每個tile在point lsit中的開始與結束位置
	if (num_rendered > 0)
		identifyTileRanges <<<(num_rendered + 255) / 256, 256 >>> (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)



	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, 
		height,
		geomState.means2D,
		feature_ptr, // rgb
		semantic_feature, // semantic
		score_feature, // score
		geomState.depths, // depth
		geomState.conic_opacity, // conic + opacity
		imgState.accum_alpha, // output
		imgState.n_contrib, // output
		background,
		out_color, // output
		out_feature_map, // output
		out_score_map, 
		out_depth // output
	), debug)

	return num_rendered;
}









// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, 
	int R, // num_rendered
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* semantic_feature, 
	const float* score_feature, // score
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix, // input color
	const float* dL_dfeaturepix, // input semantic
	const float* dL_dscorepix, // input score
	const float* dL_depths, // input
	float* dL_dmean2D,
	float* dL_dconic,  // not in final output
	float* dL_dopacity,
	float* dL_dcolor,  // precompute color
	float* dL_dsemantic_feature, // semantic
	float* dL_dscore_feature, // score
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	float* dL_dz, // depth, not in final output
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;

	// block based semantic feature size
	float* collected_semantic_feature; 
	hipMalloc((void**)&collected_semantic_feature, NUM_SEMANTIC_CHANNELS * BLOCK_SIZE * sizeof(float)); 

	// gaussian render image depth,
	const float* depth_ptr = geomState.depths;


	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,  // sorted gaussian list
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		semantic_feature,
		score_feature,
		depth_ptr, 
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,               // input
		dL_dfeaturepix, // input
		dL_dscorepix, // input
		dL_depths,                        // input
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor, // color
		dL_dsemantic_feature,  // semantic
		dL_dscore_feature, // score
		dL_dz, 				   // depth
		collected_semantic_feature
		), debug) 
		hipFree(collected_semantic_feature);



	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		dL_dz), debug)
}







